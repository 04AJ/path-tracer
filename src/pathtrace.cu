#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene *hst_scene = NULL;
static glm::vec3 *dev_image = NULL;
static Geom * dev_geom;
static Material * dev_material;
static Path * dev_path;
// TODO: static variables for device memory, any extra info you need, etc
// ...

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_path, pixelcount * sizeof(Path));

	hipMalloc(&dev_geom, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geom, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_material, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_material, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);
    // TODO: initialize any extra device memeory you need

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_path);
	hipFree(dev_geom);
	hipFree(dev_material);
    // TODO: clean up any extra device memory you created

    checkCUDAError("pathtraceFree");
}

/**
 * Example function to generate static and test the CUDA-GL interop.
 * Delete this once you're done looking at it!
 */
__global__ void generateNoiseDeleteMe(Camera cam, int iter, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);

        // CHECKITOUT: Note that on every iteration, noise gets added onto
        // the image (not replaced). As a result, the image smooths out over
        // time, since the output image is the contents of this array divided
        // by the number of iterations.
        //
        // Your renderer will do the same thing, and, over time, it will become
        // smoother.
        image[index] += glm::vec3(u01(rng));
    }
}

/**
* Generate Rays from camera through screen to the field
* which is the first generation of rays
*
* Antialiasing - num of rays per pixel
* motion blur - jitter scene position
* lens effect - jitter camera position
*/
__global__ void generateRayFromCamera(Camera cam, int iter, Path* paths)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		Path & path = paths[index];
		//getCameraRayAtPixel(path, cam, x, y, iter, index);
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(0, 1);

		path.ray.origin = cam.position;

		path.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + u01(rng))  	//u01(rng) is for jiitering for antialiasing
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + u01(rng)) 		//u01(rng) is for jiitering for antialiasing
			);

		path.pixelIndex = index;
		path.color = glm::vec3(1.0f);
		path.terminated = false;
	}
}






__global__ void pathTraceOneBounce(int depth, int num_paths, glm::vec3 * image
	, Path * paths
	, Geom * geoms, int geoms_size
	, Material * materials, int materials_size
	)
{
	//int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	//int path_index = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	//int path_index = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		Path & path = paths[path_index];	//TODO: reconsider the speed for the memory access here

		//calculate intersection
		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		//naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			//Geom & geom = static_cast<Geom>(*it);
			glm::vec3 tmp_intersect;
			glm::vec3 tmp_normal;
			Geom & geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, path.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, path.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more primitive types intersection test here

			if (t > 0 && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}


		///////////////////////////////


		if (hit_geom_index == -1)
		{
			path.terminated = true;
			//image[path.pixelIndex] += BACKGROUND_COLOR;
		}
		else
		{
			//hit something
			Geom & geom = geoms[hit_geom_index];
			Material & material = materials[geom.materialid];

			// TODO: Delete me
			// This is a test implementation, color the pixel with the hitting material value

			// TODO: call scatterRay
			// scatterRay(path.ray, path.color, intersect_point, normal, material, rng);

			image[path.pixelIndex] += material.color;

		}



	}
}













/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;
	const dim3 blockSize(blockSize1d);

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray is a (ray, color) pair, where color starts as the
    //     multiplicative identity, white = (1, 1, 1).
    //   * For debugging, you can output your ray directions as colors.
    // * For each depth:
    //   * Compute one new (ray, color) pair along each path (using scatterRay).
    //     Note that many rays will terminate by hitting a light or hitting
    //     nothing at all. You'll have to decide how to represent your path rays
    //     and how you'll mark terminated rays.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //       surface.
    //     * You can debug your ray-scene intersections by displaying various
    //       values as colors, e.g., the first surface normal, the first bounced
    //       ray direction, the first unlit material color, etc.
    //   * Add all of the terminated rays' results into the appropriate pixels.
    //   * Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    // * Finally, handle all of the paths that still haven't terminated.
    //   (Easy way is to make them black or background-colored.)

    // TODO: perform one iteration of path tracing

    //generateNoiseDeleteMe<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, dev_image);

	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> >(cam, iter, dev_path);
	checkCUDAError("generate camera ray");

	int depth = 0;
	Path* dev_path_end = dev_path + pixelcount;
	int num_path = dev_path_end - dev_path;

	// TODO: iterate your path tracing process

	dim3 blocksNeeded = (num_path + blockSize1d - 1) / blockSize1d;
	pathTraceOneBounce << <blocksNeeded, blockSize1d >> >(depth, num_path, dev_image, dev_path
		, dev_geom, hst_scene->geoms.size()
		, dev_material, hst_scene->materials.size());
	checkCUDAError("trace one bounce");
	hipDeviceSynchronize();
	depth++;





    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
